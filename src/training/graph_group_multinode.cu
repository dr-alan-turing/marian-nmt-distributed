#include "hip/hip_runtime.h"
#include "training/graph_group_multinode.h"

#include "kernels/tensor_operators.h"

namespace marian {

void MultiNodeGraphGroup::setScheduler(Ptr<Scheduler> scheduler) {
  scheduler_ = scheduler;
  // optimizer has to be registered last to see a change of learning rate
  scheduler_->registerTrainingObserver(scheduler_);

  for (auto opt : gpuShardsOpts_) {
    scheduler_->registerTrainingObserver(opt);
  }
}

Tensor MultiNodeGraphGroup::newTensor(int size, int device) {
  Tensor t;
  Ptr<TensorAllocator> allocator = New<TensorAllocator>(device);
  allocator->reserveExact(size * sizeof(float));
  allocator->allocate(t, {1, size});
  allocators_.push_back(allocator);
  return t;
}

void MultiNodeGraphGroup::initFirstRun(Ptr<data::Batch> batch) {
  // Initialize client graphs (incl. params) and builders
  for (size_t i = 0; i < graphs_.size(); ++i) {
    THREAD_GUARD(
        builders_[i]->build(graphs_[i], batch);
        graphs_[i]->forward();
    );
  }
  hipStreamSynchronize(0);
  // Initialize variables for server shard(s) on this node
  initServerShards();
  // Initialize client variables for inter-node communication
  initClientCommunicationVars();
  // Launch server thread to communicate with clients
  launchServerThread();
  // Launch compute/communicate overlap threads if enabled
  if (commOverlap_) {
    launchCommOverlapThreads();
  }
}

void MultiNodeGraphGroup::initMPI() {
#if MPI_FOUND
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_comm_world_size_);
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_my_rank_);
#endif
}

void MultiNodeGraphGroup::initServerShards(bool initFullSendReceiveBuffer) {
  // Initialize server shard sizes for all nodes (remote + current)
  size_t totalParamsGradsSize = graphs_[0]->params()->vals()->size();
  size_t nodeShardSize = ceilf(((float) totalParamsGradsSize) / mpi_comm_world_size_);
  size_t remainingTotalSize = totalParamsGradsSize;
  for (int node = 0; node < mpi_comm_world_size_; node++) {
    size_t size = std::min(nodeShardSize, remainingTotalSize);
    nodeShardSizes_.push_back(size);
    remainingTotalSize -= size;
  }

  // Initialize this shard's params and grads
  size_t thisNodeSize = nodeShardSizes_[mpi_my_rank_];
  size_t gpuShardSize = ceilf(((float) thisNodeSize) / devices_.size());
  size_t offset = 0;

  for (int gpu = 0; gpu < devices_.size(); gpu++) {
    size_t size = std::min(gpuShardSize, thisNodeSize - offset);
    Tensor gpuParams = newTensor(size, devices_[gpu]);
    gpuParams->copyFrom(graphs_[0]->params()->vals()->subtensor(offset, size));
    gpuShardsParams_.push_back(gpuParams);
    gpuShardsGrads_.push_back(newTensor(size, devices_[gpu]));
    localSubShardSizes_.push_back(size);
    offset += size;
  }

  // Initialize full send/receive buffer
  if (initFullSendReceiveBuffer) {
    serverShardBuffer_ = std::vector<float>(nodeShardSizes_[mpi_my_rank_]);
  }
}

void MultiNodeGraphGroup::setupClientsOfNodesAndDevices(std::vector<int> multiNodeDevices) {
  int index = 0, node = 0, nClientsSeen = 0;
  numberClientsOfNodes_ = std::vector<int>(mpi_comm_world_size_, 0);
  while (index < multiNodeDevices.size()) {
    if (numberClientsOfNodes_[node] == 0) {
      numberClientsOfNodes_[node] = (size_t) multiNodeDevices[index];
      nClientsSeen = 0;
    } else if (nClientsSeen < numberClientsOfNodes_[node]) {
      if (node == mpi_my_rank_) {
        devices_.push_back((size_t)multiNodeDevices[index]);
      }
      nClientsSeen++;
    } else {
      node++;
      index--;
    }
    index++;
  }
}

void MultiNodeGraphGroup::initClientCommunicationVars(bool initBuffers) { // @TODO: Integrate with clients / drop-rate / comm-overlap
  for (int gpu = 0; gpu < devices_.size(); gpu++) {
    if (initBuffers) {
      size_t size = nodeShardSizes_[mpi_my_rank_];
      clientCommBuffersCPU_.push_back(std::vector<float>(size));
    }
    if (commOverlap_) {
      size_t fullSize = graphs_[0]->params()->vals()->size();
      // Running sum of gradients
      Tensor sumGrads = newTensor(fullSize, devices_[gpu]);
      Element(functional::_1 = 0, sumGrads);
      hipStreamSynchronize(0);
      clientSummedGradsGPU.push_back(sumGrads);
      // Communication overlap buffer (for grads + params)
      Tensor commBuffer = newTensor(fullSize, devices_[gpu]);
      commBuffer->copyFrom(graphs_[0]->params()->vals());
      clientCommOverlapBuffersGPU_.push_back(commBuffer);
    }
  }
}

void MultiNodeGraphGroup::launchServerThread() {
#if MPI_FOUND
  serverShardThread_ = new std::thread([this] {
    int nCommunicatingNodes = mpi_comm_world_size_; // keep track of number of nodes still communicating with this shard
    MPI_Status status;
    do {
      // Receive grads from any client
      unsigned long messageInfo[4];
      MPI_Recv(&messageInfo, 4, MPI_UNSIGNED_LONG, MPI_ANY_SOURCE, MPI_TAG_GRAD_PUSH_, MPI_COMM_WORLD, &status);
      if (messageInfo[MSG_INFO_STATUS_] == STATUS_NODE_FINISHED_) {
        nCommunicatingNodes--;
        continue;
      } // register finished node and skip to next loop iteration
      MPI_Recv(serverShardBuffer_.data(), nodeShardSizes_[mpi_my_rank_], MPI_FLOAT, status.MPI_SOURCE, MPI_TAG_GRAD_PUSH_, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

      // Update shard params asynchronously over GPUs
      std::vector<std::thread> threads;
      size_t offset = 0;
      for (int gpu = 0; gpu < devices_.size(); gpu++) {
        size_t size = localSubShardSizes_[gpu];

        threads.emplace_back(std::thread([=](int gpu, size_t offset, size_t size, size_t batchWords) {
          std::lock_guard<std::mutex> guard(mutexGpuShards_[gpu]);

          // Copy grads to appropriate GPU
          hipMemcpy(gpuShardsGrads_[gpu]->data(), &serverShardBuffer_.at(offset), size * sizeof(float), hipMemcpyHostToDevice);
          hipStreamSynchronize(0);

          // Run optimizer on GPU
          if (scaleLearningRate_ && batchWords > 0) {
            gpuShardsOpts_[gpu]->update(gpuShardsParams_[gpu], gpuShardsGrads_[gpu], batchWords / avgBatchWords_);
          } else {
            gpuShardsOpts_[gpu]->update(gpuShardsParams_[gpu], gpuShardsGrads_[gpu]);
          }
          hipStreamSynchronize(0);
          // Copy params from GPU
          hipMemcpy(&serverShardBuffer_.at(offset), gpuShardsParams_[gpu]->data(), size * sizeof(float), hipMemcpyDeviceToHost);
          hipStreamSynchronize(0);
        }, gpu, offset, size, messageInfo[MSG_INFO_BATCHWORDS_]));

        offset += size;
      }
      for (auto &&t : threads) { t.join(); }

      // Send updated params to same client
      MPI_Ssend(serverShardBuffer_.data(), nodeShardSizes_[mpi_my_rank_], MPI_FLOAT, status.MPI_SOURCE,
                MPI_TAG_PARAM_PUSH_, MPI_COMM_WORLD);

    } while (nCommunicatingNodes != 0);
  });
#endif
}

void MultiNodeGraphGroup::synchronizeWithServerShards(Tensor newGrads, Tensor oldParams, int gpu, size_t batchWords) {
  #if MPI_FOUND
  size_t offset = 0;
  for (int node = 0; node < mpi_comm_world_size_; node++) {
    size_t nodeSize = nodeShardSizes_[node];

    // Update remotely if node != this node
    if (node != mpi_my_rank_) {

      // Copy grads from GPU to CPU (for MPI sending)
      hipMemcpy(clientCommBuffersCPU_[gpu].data(), newGrads->subtensor(offset, nodeSize)->data(), nodeSize * sizeof(float), hipMemcpyDeviceToHost);
      hipStreamSynchronize(0);

      // Send grads to server node
      size_t messageInfo[4];
      messageInfo[MSG_INFO_SIZE_] = nodeSize;
      messageInfo[MSG_INFO_CLIENT_] = gpu;
      messageInfo[MSG_INFO_BATCHWORDS_] = batchWords;
      messageInfo[MSG_INFO_STATUS_] = STATUS_NODE_TRAINING_;
      MPI_Ssend(&messageInfo, 4, MPI_UNSIGNED_LONG, node, MPI_TAG_GRAD_PUSH_, MPI_COMM_WORLD);
      MPI_Ssend(clientCommBuffersCPU_[gpu].data(), nodeSize, MPI_FLOAT, node, MPI_TAG_GRAD_PUSH_, MPI_COMM_WORLD);

      // Receive updated params from server node
      MPI_Recv(clientCommBuffersCPU_[gpu].data(), nodeSize, MPI_FLOAT, node, MPI_TAG_PARAM_PUSH_, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

      // Copy params from CPU back to GPU
      hipMemcpy(oldParams->subtensor(offset, nodeSize)->data(), clientCommBuffersCPU_[gpu].data(), nodeSize * sizeof(float), hipMemcpyHostToDevice);
      hipStreamSynchronize(0);


    // Else update locally if node == this node
    } else {
      size_t localOffset = offset;
      std::vector<std::thread> threads;

      for (int gpu = 0; gpu < devices_.size(); gpu++) {
        size_t gpuSize = localSubShardSizes_[gpu];

        threads.emplace_back(std::thread([=](int gpu, size_t offset, size_t size) {
          std::lock_guard<std::mutex> guard(mutexGpuShards_[gpu]);

          // Copy grads to appropriate GPU
          gpuShardsGrads_[gpu]->copyFrom(newGrads->subtensor(offset, size));
          // Run optimizer on GPU
          if (scaleLearningRate_ && batchWords > 0) {
            gpuShardsOpts_[gpu]->update(gpuShardsParams_[gpu], gpuShardsGrads_[gpu], batchWords / avgBatchWords_);
          } else {
            gpuShardsOpts_[gpu]->update(gpuShardsParams_[gpu], gpuShardsGrads_[gpu]);
          }
          hipStreamSynchronize(0);
          // Copy params back to current GPU
          oldParams->subtensor(offset, size)->copyFrom(gpuShardsParams_[gpu]);
        }, gpu, localOffset, gpuSize));

        localOffset += gpuSize;
      }
      for (auto &&t : threads) { t.join(); }
    }

    offset += nodeSize;
  }
  #endif
}

void MultiNodeGraphGroup::launchCommOverlapThreads() {
#if MPI_FOUND
  for (int gpu = 0; gpu < devices_.size(); gpu++) {
    clientCommThreads_.emplace_back(new std::thread([this](int gpu) {
      do {
        // Wait for GPU (client) to fill buffers pointers
        std::unique_lock<std::mutex> uniqueLock(mutexClientCommOverlapBuffersFilled_[gpu]);
        while (!clientCommOverlapBuffersFilled_[gpu]) {
          cvClientCommOverlapBuffersFilled_[gpu].wait(uniqueLock);
        }

        if (stopClientCommThreads_) { break; }

        // Synchronize with server shards
        synchronizeWithServerShards(clientCommOverlapBuffersGPU_[gpu], clientCommOverlapBuffersGPU_[gpu], gpu, scaleLearningRate_ ? clientCommittedWordCounts_[gpu] : 0);

        // Indicate that buffers can be read from and filled again
        clientCommOverlapBuffersFilled_[gpu] = false;

      } while (!stopClientCommThreads_);
    }, gpu));
  }
#endif
}

void MultiNodeGraphGroup::execute(Ptr<data::Batch> batch) {
  if (!firstBatchProcessed_) {
    initFirstRun(batch);
    firstBatchProcessed_ = true;
  }

  auto task = [this](Ptr<data::Batch> batch) {
    static size_t i = 0;
    thread_local Ptr<ExpressionGraph> graph;
    thread_local Ptr<models::ModelBase> builder;
    thread_local size_t my_id = 0;

    if (!graph) {
      std::lock_guard<std::mutex> lock(mutexClientInit_);
      my_id = i;
      graph = graphs_[i];
      builder = builders_[i++];
    }

    auto costNode = builder->build(graph, batch);

    graph->forward();
    float cost = costNode->scalar();
    graph->backward();

    hipStreamSynchronize(0);

    if(!commOverlap_) {
      synchronizeWithServerShards(graph->params()->grads(), graph->params()->vals(), my_id, batch->words());
    }

    if (scheduler_) {
      boost::upgrade_lock<boost::shared_mutex> lock(schedulerMutex_);
      {
        boost::upgrade_to_unique_lock<boost::shared_mutex> uniqueLock(lock);
        scheduler_->update(cost, batch);
      }

      if (scheduler_->saving()) {
        boost::upgrade_to_unique_lock<boost::shared_mutex> uniqueLock(lock);
        //if(movingAvg_)
        //  fetchParams(graph->params()->vals(), paramsAvg_);
        this->save(graph);
      }

      if (scheduler_->validating()) {
        boost::upgrade_to_unique_lock<boost::shared_mutex> uniqueLock(lock);
        //if(movingAvg_)
        //  fetchParams(graph->params()->vals(), paramsAvg_);
        scheduler_->validate(graphs_);
      }
    }

    // Overlapping computations with communication
    if (commOverlap_) {

      // Add computed gradients to local running sum
      Element(functional::_1 = functional::_1 + functional::_2, clientSummedGradsGPU[my_id], graph->params()->grads());
      hipStreamSynchronize(0);
      // Sum up word counts if batch flexible learning rate is enabled
      if (scaleLearningRate_) {
        clientSummedWordCounts_[my_id] += batch->words();
      }

      // If communication channel ready, swap graph's pointers with secondary buffers
      if (!clientCommOverlapBuffersFilled_[my_id]) {
        std::unique_lock<std::mutex> tryLock(mutexClientCommOverlapBuffersFilled_[my_id], std::try_to_lock);
        if (tryLock.owns_lock()) {
          // Copy parameters from communication buffer
          graph->params()->vals()->copyFrom(clientCommOverlapBuffersGPU_[my_id]);
          // Copy summed grads to communication buffer
          clientCommOverlapBuffersGPU_[my_id]->copyFrom(clientSummedGradsGPU[my_id]);
          // Commit summed word counts if batch-flexible-lr enabled
          if (scaleLearningRate_) {
            clientCommittedWordCounts_[my_id] = clientSummedWordCounts_[my_id];
            clientSummedWordCounts_[my_id] = 0;
          }
          // Notify communication thread that buffers have been read and filled
          clientCommOverlapBuffersFilled_[my_id] = true;
          cvClientCommOverlapBuffersFilled_[my_id].notify_one();
          // Apply summed gradients to new parameters
          clientLocalOpts_[my_id]->update(graph->params()->vals(), clientSummedGradsGPU[my_id]);
          // Clear summed gradients
          clientSummedGradsGPU[my_id]->set(0);
        }

      }

    }

  };

  pool_->enqueue(task, batch);
}

void MultiNodeGraphGroup::signalFinishedToServerShards() {
  #if MPI_FOUND
  unsigned long messageInfo[4];
  messageInfo[MSG_INFO_STATUS_] = STATUS_NODE_FINISHED_;
  for (int node = 0; node < mpi_comm_world_size_; node++) {
    MPI_Ssend(&messageInfo, 4, MPI_UNSIGNED_LONG, node, MPI_TAG_GRAD_PUSH_, MPI_COMM_WORLD);
  }
  #endif
}

void MultiNodeGraphGroup::shutDownServerShardThread() {
  serverShardThread_->join(); // Wait for server thread to finish communicating (with unfinished nodes)
}

void MultiNodeGraphGroup::shutDownCommOverlapThreads() {
  stopClientCommThreads_ = true;
  for (int gpu = 0; gpu < devices_.size(); gpu++) {
    clientCommOverlapBuffersFilled_[gpu] = true;
    cvClientCommOverlapBuffersFilled_[gpu].notify_one(); // Unblock thread from lock, then join it
    clientCommThreads_[gpu]->join();
  }
}

}